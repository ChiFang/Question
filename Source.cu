#include "hip/hip_runtime.h"


#include<stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

//���ޥΨ쪺���c��
struct Index {
	int block, thread;
};


//�֤�:����޼g�J�˸m�O����
__global__ void prob_idx(Index id[])
{
	int b = blockIdx.x;       //�϶�����
	int t = threadIdx.x;      //���������
	int n = blockDim.x;       //�϶����]�t��������ƥ�
	int x = b*n + t;            //������b�}�C����������m

								//�C�Ӱ�����g�J�ۤv���϶��M���������.
	id[x].block = b;
	id[x].thread = t;
};

//�D�禡
int main() {
	Index* d;
	Index  h[100];

	//�t�m�˸m�O����
	hipMalloc((void**)&d, 100 * sizeof(Index));

	//�I�s�˸m�֤�
	int g = 3, b = 4, m = g*b;
	// prob_idx<<< g, b>>>(d);

	prob_idx KERNEL_ARGS2(dim3(nBlockCount), dim3(nThreadCount)) (d);

	//�U���˸m�O���餺�e��D���W
	hipMemcpy(h, d, 100 * sizeof(Index), hipMemcpyDeviceToHost);

	//��ܤ��e
	for (int i = 0; i<m; i++) {
		printf("h[%d]={block:%d, thread:%d}\n", i, h[i].block, h[i].thread);
	}

	//����˸m�O����
	hipFree(d);
	return 0;
}